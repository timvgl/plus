#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "datatypes.hpp"
#include "field.hpp"
#include "ncafm.hpp"
#include "octupole.hpp"

__device__ real3 rotate_120(real3 m, real3 ref) {
  // rotate m about k over 120 degrees
  real3 k = normalized(cross(ref, m));
  real dir = (dot(ref, m) < 0) ? -1.0 : 1.0;  // Use 1 if perpendicular
  return -0.5 * m + cross(k, m) * 0.86602540378 * dir; // Use sqrt(3) / 2 = 0.86602540378
}

__global__ void k_octupolevector(CuField octupole,
                             const CuField mag1,
                             const CuField mag2,
                             const CuField mag3,
                             const CuParameter msat1,
                             const CuParameter msat2,
                             const CuParameter msat3) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // When outside the geometry, set to zero and return early
  if (!octupole.cellInGeometry(idx)) {
    if (octupole.cellInGrid(idx)) 
      octupole.setVectorInCell(idx, real3{0, 0, 0});
    return;
  }

  real3 m1 = mag1.vectorAt(idx);
  real3 m2 = mag2.vectorAt(idx);
  real3 m3 = mag3.vectorAt(idx);
  real ms1 = msat1.valueAt(idx);
  real ms2 = msat2.valueAt(idx);
  real ms3 = msat3.valueAt(idx);

  real3 m2_rot = rotate_120(m2, m1);
  real3 m3_rot = rotate_120(m3, m1);

  octupole.setVectorInCell(idx, (m1 * ms1 + m2_rot * ms2 + m3_rot * ms3) / (ms1 + ms2 + ms3));
}

Field evalOctupoleVector(const NcAfm* magnet) {
  // Calculate a weighted octupole vector (dimensionless) to account for NC-ferrimagnets
  Field octupole(magnet->system(), 3);

  if (magnet->sub1()->msat.assuredZero() &&
      magnet->sub2()->msat.assuredZero() &&
      magnet->sub3()->msat.assuredZero()) {
        octupole.makeZero();
        return octupole;
  }
  hipLaunchByPtr("octopole.cu", octupole.grid().ncells(), k_octupolevector, octupole.cu(),
             magnet->sub1()->magnetization()->field().cu(),
             magnet->sub2()->magnetization()->field().cu(),
             magnet->sub3()->magnetization()->field().cu(),
             magnet->sub1()->msat.cu(),
             magnet->sub2()->msat.cu(),
             magnet->sub3()->msat.cu());
  return octupole;
}

NcAfm_FieldQuantity octupoleVectorQuantity(const NcAfm* magnet) {
    return NcAfm_FieldQuantity(magnet, evalOctupoleVector, 3, "octupole_vector", "");
}