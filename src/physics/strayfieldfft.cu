#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include <memory>
#include <vector>

#include "constants.hpp"
#include "cudalaunch.hpp"
#include "fieldops.hpp"
#include "magnet.hpp"
#include "quantityevaluator.hpp"
#include "field.hpp"
#include "fullmag.hpp"
#include "grid.hpp"
#include "parameter.hpp"
#include "strayfieldfft.hpp"
#include "strayfieldkernel.hpp"
#include "system.hpp"

#if FP_PRECISION == SINGLE
const hipfftType FFT = HIPFFT_R2C;
const hipfftType IFFT = HIPFFT_C2R;
const auto& fftExec = hipfftExecR2C;
const auto& ifftExec = hipfftExecC2R;
#elif FP_PRECISION == DOUBLE
const hipfftType FFT = HIPFFT_D2Z;
const hipfftType IFFT = HIPFFT_Z2D;
const auto& fftExec = hipfftExecD2Z;
const auto& ifftExec = hipfftExecZ2D;
#endif

#define __CUDAOP__ inline __device__ __host__

// No simpel operator overloading due to definition of real2.
__CUDAOP__ complex sum(complex a, complex b) {
#if FP_PRECISION == SINGLE
  return hipCaddf(a, b);
#elif FP_PRECISION == DOUBLE
  return hipCadd(a, b);
#endif
}

__CUDAOP__ complex prod(complex a, complex b) {
#if FP_PRECISION == SINGLE
  return hipCmulf(a, b);
#elif FP_PRECISION == DOUBLE
  return hipCmul(a, b);
#endif
}

__global__ void k_pad(CuField out,
                      CuField in,
                      CuParameter msat) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;
  
  Grid outgrid = out.system.grid;
  Grid ingrid = in.system.grid;

  if (outIdx >= outgrid.ncells())
    return;

  int3 outCoo = outgrid.index2coord(outIdx);
  int3 inCoo = outCoo - outgrid.origin() + ingrid.origin();
  int inIdx = ingrid.coord2index(inCoo);

  if (in.cellInGeometry(inCoo)) {
    real Ms = msat.valueAt(inIdx);
    for (int c = 0; c < out.ncomp; c++)
      out.setValueInCell(outIdx, c, Ms * in.valueAt(inIdx, c));
  }
  else {
    for (int c = 0; c < out.ncomp; c++)
      out.setValueInCell(outIdx, c, 0.0);
  }
}


__global__ void k_unpad(CuField out, CuField in) {
  int outIdx = blockIdx.x * blockDim.x + threadIdx.x;

  // When outside the geometry of destiny field, set to zero and return
  // early
  if (!out.cellInGeometry(outIdx)) {
    if (out.cellInGrid(outIdx))
        out.setVectorInCell(outIdx, real3{0, 0, 0});
    return;
  }

  Grid outgrid = out.system.grid;
  Grid ingrid = in.system.grid;

  // Output coordinate relative to the origin of the output grid
  int3 outRelCoo = outgrid.index2coord(outIdx) - outgrid.origin();

  // Input coordinate relative to the origin of the input grid
  int3 inRelCoo = ingrid.size() - outgrid.size() + outRelCoo;

  int inIdx = ingrid.coord2index(inRelCoo + ingrid.origin());

  for (int c = 0; c < out.ncomp; c++) {
    out.setValueInCell(outIdx, c, in.valueAt(inIdx, c));
  }
}

static void checkCufftResult(hipfftResult result) {
  if (result != HIPFFT_SUCCESS)
    throw std::runtime_error("cufft error in demag convolution");
}

__global__ void k_apply_kernel_3d(complex* hx,
                                  complex* hy,
                                  complex* hz,
                                  complex* mx,
                                  complex* my,
                                  complex* mz,
                                  complex* kxx,
                                  complex* kyy,
                                  complex* kzz,
                                  complex* kxy,
                                  complex* kxz,
                                  complex* kyz,
                                  complex preFactor,
                                  int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  hx[i] = prod(preFactor, (sum(sum(prod(kxx[i], mx[i]), prod(kxy[i], my[i])), prod(kxz[i], mz[i]))));
  hy[i] = prod(preFactor, (sum(sum(prod(kxy[i], mx[i]), prod(kyy[i], my[i])), prod(kyz[i], mz[i]))));
  hz[i] = prod(preFactor, (sum(sum(prod(kxz[i], mx[i]), prod(kyz[i], my[i])), prod(kzz[i], mz[i]))));
}

__global__ void k_apply_kernel_2d(complex* hx,
                                  complex* hy,
                                  complex* hz,
                                  complex* mx,
                                  complex* my,
                                  complex* mz,
                                  complex* kxx,
                                  complex* kyy,
                                  complex* kzz,
                                  complex* kxy,
                                  complex preFactor,
                                  int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  hx[i] = prod(preFactor, (sum(prod(kxx[i], mx[i]), prod(kxy[i], my[i]))));
  hy[i] = prod(preFactor, (sum(prod(kxy[i], mx[i]), prod(kyy[i], my[i]))));
  hz[i] = prod(preFactor, prod(kzz[i], mz[i]));
}

StrayFieldFFTExecutor::StrayFieldFFTExecutor(
    const Magnet* magnet,
    std::shared_ptr<const System> system, int order, double eps, double switchingradius)
    : StrayFieldExecutor(magnet, system),
      kernel_(system->grid(), magnet_->grid(), magnet->world(), order, eps, switchingradius),
      kfft(6),
      hfft(3),
      mfft(3),
      stream_(getCudaStreamFFT()) {
  int3 size = kernel_.grid().size();
  fftSize = {size.x / 2 + 1, size.y, size.z};
  int ncells = fftSize.x * fftSize.y * fftSize.z;

  for (auto& p : kfft)
    hipMalloc(reinterpret_cast<void**>(&p), ncells * sizeof(complex));
  for (auto& p : mfft)
    hipMalloc(reinterpret_cast<void**>(&p), ncells * sizeof(complex));
  for (auto& p : hfft)
    hipMalloc(reinterpret_cast<void**>(&p), ncells * sizeof(complex));
  checkCufftResult(hipfftPlan3d(&forwardPlan, size.z, size.y, size.x, FFT));
  checkCufftResult(hipfftPlan3d(&backwardPlan, size.z, size.y, size.x, IFFT));
  


  hipfftSetStream(forwardPlan, stream_);
  hipfftSetStream(backwardPlan, stream_);

  for (int comp = 0; comp < 6; comp++)
    checkCufftResult(
        fftExec(forwardPlan, kernel_.field().device_ptr(comp), kfft.at(comp)));
  checkCudaError(hipPeekAtLastError());
  checkCudaError(hipStreamSynchronize(stream_));
}

StrayFieldFFTExecutor::~StrayFieldFFTExecutor() {
  checkCudaError(hipStreamSynchronize(stream_));
  for (auto p : mfft)
    hipFree(p);
  for (auto p : kfft)
    hipFree(p);
  for (auto p : hfft)
    hipFree(p);
  
  checkCufftResult(hipfftDestroy(forwardPlan));
  checkCufftResult(hipfftDestroy(backwardPlan));
}

Field StrayFieldFFTExecutor::exec() const {
  // pad m, and multiply with msat
  std::shared_ptr<System> kernelSystem =
      std::make_shared<System>(magnet_->world(), kernel_.grid());

  std::unique_ptr<Field> mpad(new Field(kernelSystem, 3, stream_));
  fenceStreamToStream(getCudaStream(), stream_);
  //checkCudaError(hipStreamSynchronize(getCudaStream()));
  if (const Ferromagnet* mag = magnet_->asFM()) {
    auto m = mag->magnetization()->field().cu();
    auto ms = mag->msat.cu(stream_);
    cudaLaunchStream(stream_, "strayfieldfft.cu", mpad->grid().ncells(), k_pad, mpad->cu(), m, ms);
    mag->msat.markLastUse(stream_);
  }
  else {
    auto hostmag = evalHMFullMagOn(magnet_->asHost(), stream_);
    auto ms = Parameter(magnet_->system(), stream_, 1.0);
    cudaLaunchStream(stream_, "strayfieldfft.cu", mpad->grid().ncells(), k_pad, mpad->cu(), hostmag.cu(), ms.cu(stream_));
    hostmag.markLastUse(stream_);
    ms.markLastUse(stream_);
  }

  // Forward fourier transforms
  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(
        fftExec(forwardPlan, mpad->device_ptr(comp), mfft.at(comp)));

  // apply kernel on m_fft
  int ncells = fftSize.x * fftSize.y * fftSize.z;
  complex preFactor{-MU0 / kernel_.grid().ncells(), 0};
  if (kernel_.grid().size().z == 1 && kernel_.grid().origin().z == 0) {
    // if the h field and m field are two dimensional AND are in the same plane
    // (kernel grid origin at z=0) then the kernel matrix has only 4 relevant
    // components and a more efficient cuda kernel can be used:
    cudaLaunchStream(stream_, "strayfieldfft.cu", ncells, k_apply_kernel_2d, hfft.at(0), hfft.at(1), hfft.at(2),
               mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
               kfft.at(2), kfft.at(3), preFactor, ncells);
  } else {
    cudaLaunchStream(stream_, "strayfieldfft.cu", ncells, k_apply_kernel_3d, hfft.at(0), hfft.at(1), hfft.at(2),
               mfft.at(0), mfft.at(1), mfft.at(2), kfft.at(0), kfft.at(1),
               kfft.at(2), kfft.at(3), kfft.at(4), kfft.at(5), preFactor,
               ncells);
  }

  for (int comp = 0; comp < 3; comp++)
    checkCufftResult(
      ifftExec(backwardPlan, hfft.at(comp), mpad->device_ptr(comp)));
  Field h(system_, 3, stream_);
  cudaLaunchStream(stream_, "strayfieldfft.cu", h.grid().ncells(), k_unpad, h.cu(), mpad->cu());
  mpad->markLastUse(stream_);
  h.markLastUse(stream_);
  return h;
}
