#include "hip/hip_runtime.h"
#include "antiferromagnet.hpp"
#include "cudalaunch.hpp"
#include "dmi.hpp" // used for Neumann BC
#include "dmitensor.hpp"
#include "energy.hpp"
#include "exchange.hpp"
#include "ferromagnet.hpp"
#include "field.hpp"
#include "inter_parameter.hpp"
#include "ncafm.hpp"
#include "parameter.hpp"
#include "reduce.hpp"
#include "world.hpp"

bool exchangeAssuredZero(const Ferromagnet* magnet) {
  return ((magnet->aex.assuredZero() && magnet->interExch.assuredZero())
        || magnet->msat.assuredZero());
}

// Independent FM lattice
__global__ void k_exchangeField(CuField hField,
                                const CuField mField,
                                const CuParameter aex,
                                const CuParameter msat,
                                const real3 w,  // w = 1/cellsize^2
                                const Grid mastergrid,
                                bool openBC,
                                const CuDmiTensor dmiTensor,
                                const CuInterParameter interEx,
                                const CuInterParameter scaleEx) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const auto system = hField.system;

  // When outside the geometry, set to zero and return early
  if (!hField.cellInGeometry(idx)) {
    if (hField.cellInGrid(idx)) {
      hField.setVectorInCell(idx, real3{0, 0, 0});
    }
    return;
  }

  const Grid grid = mField.system.grid;
  if (!grid.cellInGrid(idx))
    return;

  if (msat.valueAt(idx) == 0) {
    hField.setVectorInCell(idx, real3{0, 0, 0});
    return;
  }

  const int3 coo = grid.index2coord(idx);
  const real3 m = mField.vectorAt(idx);
  const real a = aex.valueAt(idx);
  
  // accumulate exchange field in h for cell at idx, divide by msat at the end
  real3 h{0, 0, 0};

  // FM exchange in NN cells
#pragma unroll
  for (int3 rel_coo : {int3{-1, 0, 0}, int3{1, 0, 0}, int3{0, -1, 0},
                            int3{0, 1, 0}, int3{0, 0, -1}, int3{0, 0, 1}}) {
    const int3 coo_ = mastergrid.wrap(coo + rel_coo);
    if(!hField.cellInGeometry(coo_) && openBC)
      continue;

    const int idx_ = grid.coord2index(coo_);

    if(msat.valueAt(idx_) != 0 || !openBC) {
      real3 m_;
      real a_;
      int3 normal = rel_coo * rel_coo;

      real inter = 0;
      real scale = 1;
      real Aex;

      if(hField.cellInGeometry(coo_)) {
        m_ = mField.vectorAt(idx_);
        a_ = aex.valueAt(idx_);

        unsigned int ridx = system.getRegionIdx(idx);
        unsigned int ridx_ = system.getRegionIdx(idx_);

        if (ridx != ridx_) {
          scale = scaleEx.valueBetween(ridx, ridx_);
          inter = interEx.valueBetween(ridx, ridx_);
        }
      }
      else { // Neumann BC
        if (a == 0)
          continue;

        real3 Gamma = getGamma(dmiTensor, idx, normal, m);
        real delta = dot(rel_coo, system.cellsize);
        m_ = m + (Gamma / (2*a)) * delta;
        a_ = a;
      }

      Aex = getExchangeStiffness(inter, scale, a, a_);
      h += 2 * Aex * dot(normal, w) * (m_ - m);
    }
  }
  hField.setVectorInCell(idx, h / msat.valueAt(idx));
}

// FM sublattice
__global__ void k_exchangeField(CuField hField,
                                const CuField m1Field,
                                const CuField m2Field,
                                const CuParameter aex,
                                const CuParameter msat,
                                const real3 w,  // w = 1/cellsize^2
                                Grid mastergrid,
                                const CuDmiTensor dmiTensor,
                                const CuInterParameter interEx,
                                const CuInterParameter scaleEx) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const auto system = hField.system;

  // When outside the geometry, set to zero and return early
  if (!hField.cellInGeometry(idx)) {
    if (hField.cellInGrid(idx)) {
      hField.setVectorInCell(idx, real3{0, 0, 0});
    }
    return;
  }

  const Grid grid = m1Field.system.grid;
  if (!grid.cellInGrid(idx))
    return;

  if (msat.valueAt(idx) == 0) {
    hField.setVectorInCell(idx, real3{0, 0, 0});
    return;
  }

  const int3 coo = grid.index2coord(idx);
  const real3 m = m1Field.vectorAt(idx);
  const real a = aex.valueAt(idx);
  
  // accumulate exchange field in h for cell at idx, divide by msat at the end
  real3 h{0, 0, 0};

  // FM exchange in NN cells
#pragma unroll
  for (int3 rel_coo : {int3{-1, 0, 0}, int3{1, 0, 0}, int3{0, -1, 0},
                            int3{0, 1, 0}, int3{0, 0, -1}, int3{0, 0, 1}}) {
    const int3 coo_ = mastergrid.wrap(coo + rel_coo);
    const int idx_ = grid.coord2index(coo_);

    real3 m_;
    real a_;
    int3 normal = rel_coo * rel_coo;

    real inter = 0;
    real scale = 1;
    real Aex;

    if(hField.cellInGeometry(coo_)) {
      if (msat.valueAt(idx_) == 0)
        continue;

      m_ = m1Field.vectorAt(idx_);
      a_ = aex.valueAt(idx_);
      unsigned int ridx = system.getRegionIdx(idx);
      unsigned int ridx_ = system.getRegionIdx(idx_);
      if (ridx != ridx_) {
        scale = scaleEx.valueBetween(ridx, ridx_);
        inter = interEx.valueBetween(ridx, ridx_);
      }
    }
    else { // Neumann BC
      if (a == 0)
        continue;

      real3 m2 = m2Field.vectorAt(idx);
      real3 Gamma1 = getGamma(dmiTensor, idx, normal, m);

      real delta = dot(rel_coo, system.cellsize);

      real3 d_m2{0, 0, 0};
      int3 coo__ = mastergrid.wrap(coo - rel_coo);
      if(hField.cellInGeometry(coo__)) {
        // Approximate normal derivative of sister sublattice by taking
        // the bulk derivative closest to the edge.
        real3 m2__ = m2Field.vectorAt(coo__);
        d_m2 = (m2 - m2__) / delta;
      }
      m_ = m + (cross(cross(d_m2, m), m) + Gamma1) * delta / (2*a);
      a_ = a;
    }

    Aex = getExchangeStiffness(inter, scale, a, a_);
    h += 2 * Aex * dot(normal, w) * (m_ - m);
  }
  hField.setVectorInCell(idx, h / msat.valueAt(idx));
}

__global__ void k_effectiveSublattice(CuField netSub,
                                      const CuField mField,
                                      const CuParameter msat,
                                      const CuParameter afmex_nn,
                                      const CuInterParameter inter_nn,
                                      const CuInterParameter scale_nn,
                                      Grid mastergrid) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const auto system = netSub.system;

  if (!netSub.cellInGeometry(idx) || msat.valueAt(idx) == 0)
    return;

  const Grid grid = mField.system.grid;
  const int3 coo = grid.index2coord(idx);
  const real3 m = mField.vectorAt(idx);
  const real ann = afmex_nn.valueAt(idx);
  const unsigned int ridx = system.getRegionIdx(idx);

  real3 result = netSub.vectorAt(idx);

#pragma unroll
  for (int3 rel_coo : {int3{-1, 0, 0}, int3{1, 0, 0}, int3{0, -1, 0},
                       int3{0, 1, 0}, int3{0, 0, -1}, int3{0, 0, 1}}) {
    const int3 coo_ = mastergrid.wrap(coo - rel_coo);
    if(!netSub.cellInGeometry(coo_))
      continue;
    const int idx_ = grid.coord2index(coo_);
    const unsigned int ridx_ = system.getRegionIdx(idx_);

    real A = getExchangeStiffness(inter_nn.valueBetween(ridx, ridx_),
                                  scale_nn.valueBetween(ridx, ridx_),
                                  ann, afmex_nn.valueAt(idx_));
    result += (m * A);
  }
  netSub.setVectorInCell(idx, result);
}

Field evalEffectiveSublattice(const Ferromagnet* magnet) {
  // Calculate effective sublattice term in Neumann BC
  Field netSub(magnet->system(), 3, real3{0, 0, 0});

  auto host = magnet->hostMagnet();
  auto inter = host->interAfmExchNN.cu();
  auto scale = host->scaleAfmExchNN.cu();
  auto Ann = host->afmex_nn.cu();

  for (auto sub : host->getOtherSublattices(magnet)) {
    auto m = sub->magnetization()->field().cu();
    auto msat = sub->msat.cu();
    hipLaunchByPtr("exchange.cu", netSub.grid().ncells(), k_effectiveSublattice, netSub.cu(), m, msat,
               Ann, inter, scale, magnet->world()->mastergrid());
    sub->msat.markLastUse();
  }
  magnet->hostMagnet()->afmex_nn.markLastUse();
  magnet->hostMagnet()->interAfmExchNN.markLastUse();
  magnet->hostMagnet()->scaleAfmExchNN.markLastUse();
  netSub.markLastUse();
  return netSub;
}

Field evalExchangeField(const Ferromagnet* magnet) {

  Field hField(magnet->system(), 3);
  
  if (exchangeAssuredZero(magnet)) {
    hField.makeZero();
    return hField;
  }

  real3 c = magnet->cellsize();
  real3 w = {1 / (c.x * c.x), 1 / (c.y * c.y), 1 / (c.z * c.z)};
  
  int ncells = hField.grid().ncells();
  auto mag = magnet->magnetization()->field().cu();
  auto msat = magnet->msat.cu();
  auto aex = magnet->aex.cu();
  auto grid = magnet->world()->mastergrid();
  auto dmiTensor = magnet->dmiTensor.cu();
  auto interEx = magnet->interExch.cu();
  auto scaleEx = magnet->scaleExch.cu();

  if (!magnet->isSublattice() || magnet->enableOpenBC)
    hipLaunchByPtr("exchange.cu", ncells, k_exchangeField, hField.cu(), mag, aex, msat, w, grid,
               magnet->enableOpenBC, dmiTensor, interEx, scaleEx);
  else {
    // In case `magnet` is a sublattice, it's sister sublattice(s) affect(s)
    // the Neumann BC. There are no open boundaries when in this scope.
    auto sister = evalEffectiveSublattice(magnet);
    hipLaunchByPtr("exchange.cu", ncells, k_exchangeField, hField.cu(), mag, sister.cu(), aex,
              msat, w, grid, dmiTensor, interEx, scaleEx);
    sister.markLastUse();
  }
  magnet->msat.markLastUse();
  magnet->aex.markLastUse();
  magnet->dmiTensor.markLastUse();
  magnet->interExch.markLastUse();
  magnet->scaleExch.markLastUse();
  hField.markLastUse();
  return hField;
}

Field evalExchangeEnergyDensity(const Ferromagnet* magnet) {
  if (exchangeAssuredZero(magnet))
    return Field(magnet->system(), 1, 0.0);
  Field hex = evalExchangeField(magnet);
  Field eex = evalEnergyDensity(magnet, hex, 0.5);
  hex.markLastUse();
  eex.markLastUse();
  return eex;
}

real evalExchangeEnergy(const Ferromagnet* magnet) {
  if (exchangeAssuredZero(magnet))
    return 0;

  real edens = exchangeEnergyDensityQuantity(magnet).average()[0];
  return energyFromEnergyDensity(magnet, edens);
}

FM_FieldQuantity exchangeFieldQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalExchangeField, 3, "exchange_field", "T");
}

FM_FieldQuantity exchangeEnergyDensityQuantity(const Ferromagnet* magnet) {
  return FM_FieldQuantity(magnet, evalExchangeEnergyDensity, 1,
                          "exchange_energy_density", "J/m3");
}

FM_ScalarQuantity exchangeEnergyQuantity(const Ferromagnet* magnet) {
  return FM_ScalarQuantity(magnet, evalExchangeEnergy, "exchange_energy", "J");
}

__global__ void k_maxangle(CuField maxAngleField,
                           const CuField mField,
                           const CuParameter aex,
                           const CuParameter msat,
                           const Grid mastergrid) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // When outside the geometry, set to zero and return early
  if (!maxAngleField.cellInGeometry(idx)) {
    if (maxAngleField.cellInGrid(idx)) 
      maxAngleField.setValueInCell(idx, 0, 0);
    return;
  }

  const Grid grid = maxAngleField.system.grid;

  if (msat.valueAt(idx) == 0) {
    maxAngleField.setValueInCell(idx, 0, 0);
    return;
  }

  const int3 coo = grid.index2coord(idx);
  const real a = aex.valueAt(idx);

  real maxAngle{0};  // maximum angle in this cell

  int3 neighborRelativeCoordinates[6] = {int3{-1, 0, 0}, int3{0, -1, 0},
                                         int3{0, 0, -1}, int3{1, 0, 0},
                                         int3{0, 1, 0},  int3{0, 0, 1}};

#pragma unroll
  for (int3 relcoo : neighborRelativeCoordinates) {
    const int3 coo_ = mastergrid.wrap(coo + relcoo);
    if (!mField.cellInGeometry(coo_))
      continue;
    const int idx_ = grid.coord2index(coo_);
    if (msat.valueAt(idx_) != 0) {
      real a_ = aex.valueAt(idx_);
      real3 m = mField.vectorAt(idx);
      real3 m_ = mField.vectorAt(idx_);
      real angle = m == m_ ? 0 : acos(dot(m, m_));
      if (harmonicMean(a, a_) != 0 && angle > maxAngle)
        maxAngle = angle;
    }
  }
  maxAngleField.setValueInCell(idx, 0, maxAngle);
}

real evalMaxAngle(const Ferromagnet* magnet) {
  Field maxAngleField(magnet->system(), 1);
  hipLaunchByPtr("exchange.cu", maxAngleField.grid().ncells(), k_maxangle, maxAngleField.cu(),
             magnet->magnetization()->field().cu(), magnet->aex.cu(),
             magnet->msat.cu(), magnet->world()->mastergrid());
  magnet->aex.markLastUse();
  magnet->msat.markLastUse();
  maxAngleField.markLastUse();
  return maxAbsValue(maxAngleField);
}

FM_ScalarQuantity maxAngle(const Ferromagnet* magnet) {
  return FM_ScalarQuantity(magnet, evalMaxAngle, "max_angle", "rad");
}
