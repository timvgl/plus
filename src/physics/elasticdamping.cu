#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "elasticdamping.hpp"
#include "magnet.hpp"
#include "field.hpp"
#include "parameter.hpp"


bool elasticDampingAssuredZero(const Magnet* magnet) {
    return ((!magnet->enableElastodynamics()) || magnet->eta.assuredZero());
}

// Dedicated kernel function for -1 * eta * v; otherwise need two kernel calls.
__global__ void k_elasticDamping(CuField fField,
                                 const CuField vField,
                                 const CuParameter eta) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const CuSystem system = fField.system;

  // When outside the geometry, set to zero and return early
  if (!system.inGeometry(idx)) {
    if (system.grid.cellInGrid(idx)) {
      fField.setVectorInCell(idx, real3{0, 0, 0});
    }
    return;
  }

  fField.setVectorInCell(idx, -eta.valueAt(idx) * vField.vectorAt(idx));
}

Field evalElasticDamping(const Magnet* magnet) {
    Field fField(magnet->system(), 3);
    if (elasticDampingAssuredZero(magnet)) {
        fField.makeZero();
        return fField;
    }

    int ncells = fField.grid().ncells();
    CuField vField = magnet->elasticVelocity()->field().cu();
    CuParameter eta = magnet->eta.cu();

    hipLaunchByPtr("elasticdamping.cu", ncells, k_elasticDamping, fField.cu(), vField, eta);

    return fField;
}

M_FieldQuantity elasticDampingQuantity(const Magnet* magnet) {
  return M_FieldQuantity(magnet, evalElasticDamping, 3,
                         "elastic_damping", "N/m3");
}
