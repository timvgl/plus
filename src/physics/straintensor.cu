#include "hip/hip_runtime.h"
#include "cudalaunch.hpp"
#include "magnet.hpp"
#include "field.hpp"
#include "parameter.hpp"
#include "straintensor.hpp"


bool strainTensorAssuredZero(const Magnet* magnet) {
  return !magnet->enableElastodynamics();
}


__global__ void k_strainTensor(CuField strain,
                               const CuField u,
                               const real3 w,  // w = 1/cellsize
                               const Grid mastergrid) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const CuSystem system = strain.system;
  const Grid grid = system.grid;

  // When outside the geometry, set to zero and return early
  if (!system.inGeometry(idx)) {
    if (grid.cellInGrid(idx)) {
      for (int i = 0; i < strain.ncomp; i++)
        strain.setValueInCell(idx, i, 0);
    }
    return;
  }

  const real ws[3] = {w.x, w.y, w.z};
  const int3 im2_arr[3] = {int3{-2, 0, 0}, int3{0,-2, 0}, int3{0, 0,-2}};
  const int3 im1_arr[3] = {int3{-1, 0, 0}, int3{0,-1, 0}, int3{0, 0,-1}};
  const int3 ip1_arr[3] = {int3{ 1, 0, 0}, int3{0, 1, 0}, int3{0, 0, 1}};
  const int3 ip2_arr[3] = {int3{ 2, 0, 0}, int3{0, 2, 0}, int3{0, 0, 2}};
  const int3 coo = grid.index2coord(idx);

  real der[3][3] = {{0,0,0}, {0,0,0}, {0,0,0}};  // derivatives ∂i(mj)
  real3 u_0 = u.vectorAt(idx);
#pragma unroll
  for (int i = 0; i < 3; i++) {  // i is a {x, y, z} direction
    // take translation in i direction
    real wi = ws[i]; 
    int3 im2 = im2_arr[i], im1 = im1_arr[i];  // transl in direction -i
    int3 ip1 = ip1_arr[i], ip2 = ip2_arr[i];  // transl in direction +i

    int3 coo_im2 = mastergrid.wrap(coo + im2);
    int3 coo_im1 = mastergrid.wrap(coo + im1);
    int3 coo_ip1 = mastergrid.wrap(coo + ip1);
    int3 coo_ip2 = mastergrid.wrap(coo + ip2);

    // determine a derivative ∂i(m)
    real3 dudi;
    if (!system.inGeometry(coo_im1) && !system.inGeometry(coo_ip1)) {
      // --1-- zero
      dudi = real3{0, 0, 0};
    } else if ((!system.inGeometry(coo_im2) || !system.inGeometry(coo_ip2)) &&
                system.inGeometry(coo_im1) && system.inGeometry(coo_ip1)) {
      // -111-, 1111-, -1111 central difference,  ε ~ h^2
      dudi = 0.5 * (u.vectorAt(coo_ip1) - u.vectorAt(coo_im1));
    } else if (!system.inGeometry(coo_im2) && !system.inGeometry(coo_ip1)) {
      // -11-- backward difference, ε ~ h^1
      dudi =  (u_0 - u.vectorAt(coo_im1));
    } else if (!system.inGeometry(coo_im1) && !system.inGeometry(coo_ip2)) {
      // --11- forward difference,  ε ~ h^1
      dudi = (-u_0 + u.vectorAt(coo_ip1));
    } else if (system.inGeometry(coo_im2) && !system.inGeometry(coo_ip1)) {
      // 111-- backward difference, ε ~ h^2
      dudi =  (0.5 * u.vectorAt(coo_im2) - 2.0 * u.vectorAt(coo_im1) + 1.5 * u_0);
    } else if (!system.inGeometry(coo_im1) && system.inGeometry(coo_ip1)) {
      // --111 forward difference,  ε ~ h^2
      dudi = (-0.5 * u.vectorAt(coo_ip2) + 2.0 * u.vectorAt(coo_ip1) - 1.5 * u_0);
    } else {
      // 11111 central difference,  ε ~ h^4
      dudi = ((2.0/3.0)  * (u.vectorAt(coo_ip1) - u.vectorAt(coo_im1)) + 
              (1.0/12.0) * (u.vectorAt(coo_im2) - u.vectorAt(coo_ip2)));
    }
    dudi *= wi;

    der[i][0] = dudi.x;
    der[i][1] = dudi.y;
    der[i][2] = dudi.z;
  }

  // create the strain tensor
  for (int i = 0; i < 3; i++){
    for (int j = i; j < 3; j++){
      if (i == j) {  // diagonals
        strain.setValueInCell(idx, i, der[i][j]);
      }
      else {  // off-diagonal
        strain.setValueInCell(idx, i+j+2,
                              0.5 * (der[i][j] + der[j][i]));
      }
    }
  }
}


Field evalStrainTensor(const Magnet* magnet) {
  Field strain(magnet->system(), 6);
  if (strainTensorAssuredZero(magnet)) {
    strain.makeZero();
    strain.markLastUse();
    return strain;
  }

  int ncells = strain.grid().ncells();
  CuField u = magnet->elasticDisplacement()->field().cu();
  real3 w = 1 / magnet->cellsize();
  Grid mastergrid = magnet->world()->mastergrid();

  hipLaunchByPtr("straintensor.cu", ncells, k_strainTensor, strain.cu(), u, w, mastergrid);
  strain.markLastUse();
  return strain;
}


M_FieldQuantity strainTensorQuantity(const Magnet* magnet) {
  return M_FieldQuantity(magnet, evalStrainTensor, 6, "strain_tensor", "");
}

// --------------------
// Strain Rate

Field evalStrainRate(const Magnet* magnet) {
  Field strainRate(magnet->system(), 6);  // symmetric 3x3 tensor
  if (strainTensorAssuredZero(magnet)) {  // same condition
    strainRate.makeZero();
    strainRate.markLastUse();
    return strainRate;
  }

  int ncells = strainRate.grid().ncells();
  CuField v = magnet->elasticVelocity()->field().cu();
  real3 w = 1/ magnet->cellsize();
  Grid mastergrid = magnet->world()->mastergrid();

  // The math for strain rate is exactly the same as for strain tensor,
  // but applied to velocity instead of displacement.
  hipLaunchByPtr("straintensor.cu", ncells, k_strainTensor, strainRate.cu(), v, w, mastergrid);
  strainRate.markLastUse();
  return strainRate;
}

M_FieldQuantity strainRateQuantity(const Magnet* magnet) {
  return M_FieldQuantity(magnet, evalStrainRate, 6, "strain_rate", "1/s");
}
