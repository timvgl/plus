#include "hip/hip_runtime.h"
#include <stdexcept>

#include "cudaerror.hpp"
#include "cudalaunch.hpp"
#include "cudastream.hpp"
#include "datatypes.hpp"
#include "field.hpp"
#include "gpubuffer.hpp"
#include "reduce.hpp"

__global__ void k_maxAbsValue(real* result, CuField f) {
  // Reduce to a block
  __shared__ real sdata[BLOCKDIM];
  int ncells = f.system.grid.ncells();
  int tid = threadIdx.x;
  real threadValue = 0.0;
  for (int i = tid; i < ncells; i += BLOCKDIM) {
    if (!f.cellInGeometry(i))
      continue;
    for (int c = 0; c < f.ncomp; c++) {
      real value = abs(f.valueAt(i, c));
      threadValue = value > threadValue ? value : threadValue;
    }
  }
  sdata[tid] = threadValue;
  __syncthreads();

  // Reduce the block
  for (unsigned int s = BLOCKDIM / 2; s > 0; s >>= 1) {
    if (tid < s)
      if (sdata[tid + s] > sdata[tid])
        sdata[tid] = sdata[tid + s];
    __syncthreads();
  }
  // TODO: check if loop unrolling makes sense here

  // Set the result
  if (tid == 0)
    *result = sdata[0];
}

real maxAbsValue(const Field& f) {
  GpuBuffer<real> d_result(1, f.getStream());
  cudaLaunchReductionKernel(k_maxAbsValue, d_result.get(), f.cu());

  // copy the result to the host and return
  real result;
  checkCudaError(hipMemcpyAsync(&result, d_result.get(), 1 * sizeof(real),
                                 hipMemcpyDeviceToHost, getCudaStream()));
  return result;
}

__global__ void k_maxVecNorm(real* result, CuField f) {
  // Reduce to a block
  __shared__ real sdata[BLOCKDIM];
  int ncells = f.system.grid.ncells();
  int tid = threadIdx.x;
  real threadValue = 0.0;
  for (int i = tid; i < ncells; i += BLOCKDIM) {
    if (!f.cellInGeometry(i))
      continue;

    real3 cellVec = f.vectorAt(i);
    real cellNorm = norm(cellVec);
    if (cellNorm > threadValue)
      threadValue = cellNorm;
  }
  sdata[tid] = threadValue;
  __syncthreads();

  // Reduce the block
  for (unsigned int s = BLOCKDIM / 2; s > 0; s >>= 1) {
    if (tid < s)
      if (sdata[tid + s] > sdata[tid])
        sdata[tid] = sdata[tid + s];
    __syncthreads();
  }
  // TODO: check if loop unrolling makes sense here

  // Set the result
  if (tid == 0)
    *result = sdata[0];
}

real maxVecNorm(const Field& f) {
  if (f.ncomp() != 3) {
    throw std::runtime_error(
        "the input field of maxVecNorm should have 3 components");
  }
  
  GpuBuffer<real> d_result(1, f.getStream());
  cudaLaunchReductionKernel(k_maxVecNorm, d_result.get(), f.cu());

  // copy the result to the host and return
  real result;
  checkCudaError(hipMemcpyAsync(&result, d_result.get(), 1 * sizeof(real),
                                 hipMemcpyDeviceToHost, getCudaStream()));
  return result;
}

__global__ void k_average(real* result, CuField f, int comp, int cellsingeo) {
  __shared__ real sdata[BLOCKDIM];
  int tid = threadIdx.x;
  int ncells = f.system.grid.ncells();

  // Reduce to a block
  real threadValue = 0.0;
  for (int i = tid; i < ncells; i += BLOCKDIM) {
    if (!f.cellInGeometry(i))
      continue;
    threadValue += f.valueAt(i, comp);
  }
  sdata[tid] = threadValue;
  __syncthreads();

  // Reduce the block
  for (unsigned int s = BLOCKDIM / 2; s > 0; s >>= 1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  // TODO: check if loop unrolling makes sense here

  // Set the result
  if (tid == 0)
    *result = sdata[0] / cellsingeo;
}

real fieldComponentAverage(const Field& f, int comp) {
  if (comp >= f.ncomp()) {
    throw std::runtime_error("Can not take the average of component " +
                             std::to_string(comp) +
                             " of a field which has only " +
                             std::to_string(f.ncomp()) + " components");
  }
  
  real result;
  int cellsingeo = f.system()->cellsInGeo();
  GpuBuffer<real> d_result(1, f.getStream());
  cudaLaunchReductionKernel(k_average, d_result.get(), f.cu(), comp, cellsingeo);
  checkCudaError(hipMemcpyAsync(&result, d_result.get(), sizeof(real),
                                 hipMemcpyDeviceToHost, getCudaStream()));
  return result;
}

std::vector<real> fieldAverage(const Field& f) {
  std::vector<real> result;
  for (int c = 0; c < f.ncomp(); c++)
    result.push_back(fieldComponentAverage(f, c));
  return result;
}

__global__ void k_dotSum(real* result, CuField f, CuField g) {
  __shared__ real sdata[BLOCKDIM];
  int ncells = f.system.grid.ncells();
  int tid = threadIdx.x;

  real threadValue = 0.0;
  for (int i = tid; i < ncells; i += BLOCKDIM) {
    if (!f.cellInGeometry(i))
      continue;
    for (int c = 0; c < f.ncomp; c++)
      threadValue += f.valueAt(i, c) * g.valueAt(i, c);
  }

  sdata[tid] = threadValue;
  __syncthreads();

  // Reduce the block
  for (unsigned int s = BLOCKDIM / 2; s > 0; s >>= 1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }

  // Set the result
  if (tid == 0)
    *result = sdata[0];
}

real dotSum(const Field& f, const Field& g) {
  if (f.system() != g.system())
    throw std::invalid_argument(
        "Can not take the dot sum of the two fields because they are not "
        "defined on the same system.");

  GpuBuffer<real> d_result(1, f.getStream());
  cudaLaunchReductionKernel(k_dotSum, d_result.get(), f.cu(), g.cu());

  // copy the result to the host and return
  real result;
  checkCudaError(hipMemcpyAsync(&result, d_result.get(), sizeof(real),
                                 hipMemcpyDeviceToHost, getCudaStream()));
  return result;
}

__global__ void k_idxInRegions(bool* result, unsigned int* regions, size_t size, unsigned int ridx) {
  __shared__ bool sdata[BLOCKDIM];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int thread_id = threadIdx.x;

  bool found = false;

  for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
    if (regions[i] == ridx) {
      found = true;
      break;
    }
  }

  sdata[thread_id] = found;
  __syncthreads();

  // Reduce the block
  for (unsigned int s = BLOCKDIM / 2; s > 0; s >>= 1) {
    if (thread_id < s)
      sdata[thread_id] = sdata[thread_id] || sdata[thread_id + s];
    __syncthreads();
  }

  // Set the result
  if (thread_id == 0)
    *result = sdata[0];
}

bool idxInRegions(GpuBuffer<unsigned int> regions, unsigned int idx) {

  GpuBuffer<bool> d_result(1, getCudaStream());
  cudaLaunchReductionKernel(k_idxInRegions, d_result.get(), regions.get(), regions.size(), idx);

  // copy the result to the host and return
  bool result;
  checkCudaError(hipMemcpyAsync(&result, d_result.get(), sizeof(bool),
                                 hipMemcpyDeviceToHost, getCudaStream()));
  return result;
}

__global__ void k_isUniformComponent(bool* isUniform, CuField f, int c) {
  __shared__ bool sdata[BLOCKDIM];
  int ncells = f.system.grid.ncells();
  int tid = threadIdx.x;

  bool result = true;

  real firstValue = 0.0;
  for (int i = 0; i < ncells; i ++)
    if (f.cellInGeometry(i)) {
      firstValue = f.valueAt(i, c);
      break;
    }

  for (int i = tid; i < ncells; i += BLOCKDIM) {
    if (!f.cellInGeometry(i))
      continue;

    if (f.valueAt(i, c) != firstValue) {
      result = false;
      break;
    }
  }

  sdata[tid] = result;
  __syncthreads();

  for (unsigned int s = BLOCKDIM / 2; s > 0; s >>= 1) {
    if (tid < s)
      sdata[tid] = sdata[tid] && sdata[tid + s];
    __syncthreads();
  }

  if (tid == 0)
    *isUniform = sdata[0];
}

bool isUniformFieldComponent(const Field& f, int comp) {
  GpuBuffer<bool> d_isUniform(1, f.getStream());

  cudaLaunchReductionKernel(k_isUniformComponent, d_isUniform.get(), f.cu(), comp);

  bool isUniform;
  checkCudaError(hipMemcpyAsync(&isUniform, d_isUniform.get(), sizeof(bool),
                                 hipMemcpyDeviceToHost, getCudaStream()));
  return isUniform;
}

bool isUniformField(const Field& f) {
  for (int c = 0; c < f.ncomp(); c++) {
    if (!isUniformFieldComponent(f, c))
      return false;
  }
  return true;
}