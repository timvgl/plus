#include "cudastream.hpp"
#include "cudaerror.hpp"

hipStream_t stream0;
hipStream_t streamFFT;
hipStream_t GCStream;

hipStream_t getCudaStream() {
  if (!stream0)
    checkCudaError(hipStreamCreate(&stream0));
  return stream0;
}

hipStream_t getCudaStreamFFT() {
  if (!streamFFT)
    checkCudaError(hipStreamCreate(&streamFFT));
  return streamFFT;
}

hipStream_t getCudaStreamGC() {
  if (!GCStream)
    checkCudaError(hipStreamCreateWithFlags(&GCStream, hipStreamNonBlocking));
  return GCStream;
}
