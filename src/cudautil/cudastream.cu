#include "cudastream.hpp"

hipStream_t stream0;
hipStream_t streamFFT;

hipStream_t getCudaStream() {
  if (!stream0)
    hipStreamCreate(&stream0);
  return stream0;
}

hipStream_t getCudaStreamFFT() {
  if (!streamFFT)
    hipStreamCreate(&streamFFT);
  return streamFFT;
}
